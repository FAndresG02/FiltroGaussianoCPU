#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION 
#include "C:\\Users\\andre\\Documents\\COMPUTACION_PARALELA\\Librerias\\stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "C:\\Users\\andre\\Documents\\COMPUTACION_PARALELA\\Librerias\\stb_image_write.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

#define KERNEL_SIZE 21
#define OFFSET (KERNEL_SIZE / 2)

// Genera un kernel gaussiano con tamaño y sigma especificados
void generarKernelGaussiano(float* kernel, int kernelSize, float sigma) {
    float sum = 0.0f;
    int offset = kernelSize / 2;

    // Recorre cada posición del kernel
    for (int y = -offset; y <= offset; y++) {
        for (int x = -offset; x <= offset; x++) {
            // Calcula el valor gaussiano para la posición (x, y)
            float exponent = -(x * x + y * y) / (2.0f * sigma * sigma);
            float value = expf(exponent) / (2.0f * M_PI * sigma * sigma);
            kernel[(y + offset) * kernelSize + (x + offset)] = value;
            sum += value; // Acumula la suma para normalizar
        }
    }

    // Normaliza el kernel para que la suma total sea 1
    for (int i = 0; i < kernelSize * kernelSize; i++) {
        kernel[i] /= sum;
    }
}

// Aplica el filtro gaussiano sobre una imagen en escala de grises
void aplicarFiltroGaussianoCPU(unsigned char* input, unsigned char* output, int width, int height, float* kernel) {
    // Recorre cada píxel de la imagen
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            float sum = 0.0f;

            // Aplica la convolución con el kernel gaussiano
            for (int ky = -OFFSET; ky <= OFFSET; ky++) {
                for (int kx = -OFFSET; kx <= OFFSET; kx++) {
                    // Calcula la posición del píxel vecino
                    int px = x + kx;
                    int py = y + ky;

                    // Aplica bordes reflejados para evitar acceder fuera de la imagen
                    if (px < 0) px = 0;
                    if (py < 0) py = 0;
                    if (px >= width) px = width - 1;
                    if (py >= height) py = height - 1;

                    // Obtiene el valor del píxel y su peso en el kernel
                    float pixel = (float)input[py * width + px];
                    float weight = kernel[(ky + OFFSET) * KERNEL_SIZE + (kx + OFFSET)];
                    sum += pixel * weight; // Acumula el valor ponderado
                }
            }

            // Asigna el nuevo valor al píxel en la imagen de salida
            output[y * width + x] = (unsigned char)fminf(fmaxf(sum, 0.0f), 255.0f);
        }
    }
}

int main() {
    int width, height, channels;

    // Carga una imagen en escala de grises
    unsigned char* gray = stbi_load("C:/Users/andre/Documents/TRABAJO/Tarea en clase - Filtro/img.jpg", &width, &height, &channels, 1);
    if (!gray) {
        printf("No se pudo cargar la imagen.\n");
        return -1;
    }

    // Reserva memoria para la imagen de salida
    size_t imageSize = width * height * sizeof(unsigned char);
    unsigned char* result = (unsigned char*)malloc(imageSize);
    if (!result) {
        printf("No se pudo asignar memoria para la imagen de salida.\n");
        stbi_image_free(gray);
        return -1;
    }

    // Genera el kernel gaussiano
    float h_kernel[KERNEL_SIZE * KERNEL_SIZE];
    float sigma = 3.5f; // Puedes ajustar este valor para cambiar la intensidad del desenfoque
    generarKernelGaussiano(h_kernel, KERNEL_SIZE, sigma);

    // Mide el tiempo de ejecución del filtro
    clock_t start = clock();
    aplicarFiltroGaussianoCPU(gray, result, width, height, h_kernel);
    clock_t end = clock();

    double elapsedTime = 1000.0 * (end - start) / CLOCKS_PER_SEC;

    // Guarda la imagen filtrada
    if (stbi_write_jpg("C:/Users/andre/Documents/21CPU.jpg", width, height, 1, result, 100)) {
        printf("Imagen JPG guardada correctamente.\n");
    }
    else {
        printf("Error al guardar la imagen JPG.\n");
    }

    printf("Filtro gaussiano aplicado en CPU.\n");
    printf("Tiempo total de ejecución: %.2f ms\n", elapsedTime);

    // Libera la memoria utilizada
    stbi_image_free(gray);
    free(result);

    return 0;
}
